#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <cmath>

template <typename scalar_t>
__global__ void av_fw_kernel(
    const torch::PackedTensorAccessor<scalar_t, 4, torch::RestrictPtrTraits, size_t> attn_weight,
    const torch::PackedTensorAccessor<scalar_t, 4, torch::RestrictPtrTraits, size_t> values,
    torch::PackedTensorAccessor<scalar_t, 4, torch::RestrictPtrTraits, size_t> output,
    int height,
    int width,
    int depth,
    int kernel_size
){
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < (values.size(0) * values.size(1))){
        const int y = blockIdx.y * blockDim.y + threadIdx.y;
        if (y < (height * width * depth)){
            const int z = blockIdx.z * blockDim.z + threadIdx.z;
            if (z < values.size(3)){
                const int b = x / values.size(1);
                const int h = x - b * values.size(1);
                const int i = y / (width * depth);
                const int j = (y % (width * depth)) / depth;
                const int d = y % depth;

                const int start_i = i - (kernel_size - 1) / 2;
                const int start_j = j - (kernel_size - 1) / 2;
                const int start_d = d - (kernel_size - 1) / 2;

                scalar_t updt = scalar_t(0);
                int k_offset = 0;

                #pragma unroll
                for (int current_i = start_i; current_i < (start_i + kernel_size); ++current_i){
                    #pragma unroll
                    for (int current_j = start_j; current_j < (start_j + kernel_size); ++current_j){
                        #pragma unroll
                        for (int current_d = start_d; current_d < (start_d + kernel_size); ++current_d){
                            if ((current_i >= 0) && (current_i < height) &&
                                (current_j >= 0) && (current_j < width) &&
                                (current_d >= 0) && (current_d < depth)){
                                const int current_offset = current_i * width * depth + current_j * depth + current_d;
                                updt += attn_weight[b][h][y][k_offset] * values[b][h][current_offset][z]; 
                            }
                            ++k_offset;
                        }
                    }
                }
                output[b][h][y][z] = updt; 
            }
        }
    }
}

torch::Tensor av_fw_cu(
    const torch::Tensor attn_weight,
    const torch::Tensor values,
    int height,
    int width,
    int depth,
    int kernel_size,
    int cuda_threads
){
    TORCH_CHECK((cuda_threads > 0) && (cuda_threads <= 1024), "The value of CUDA_NUM_THREADS should be between 1 and 1024");
    TORCH_CHECK(attn_weight.size(0) == values.size(0), "Attention Weights and Values should have the same Batch Size");
    TORCH_CHECK(attn_weight.size(1) == values.size(1), "Attention Weights and Values should have the same Number of Heads");
    TORCH_CHECK(attn_weight.size(2) == values.size(2), "Attention Weights and Values should have the same Pixel Numbers");

    const int B = values.size(0), N = values.size(1), L = values.size(2), C = values.size(3);

    const int DIMTHREADS = min(cuda_threads, C);
    const int PIXELTHREADS = min(int(cuda_threads / DIMTHREADS), L);
    const int BATCHTHREADS = max(1, cuda_threads / (PIXELTHREADS * DIMTHREADS));
    
    torch::Tensor output = torch::empty({B, N, L, C}, attn_weight.options());

    const dim3 threads(BATCHTHREADS, PIXELTHREADS, DIMTHREADS);
    const dim3 blocks(((B * N) + threads.x - 1) / threads.x, ((height * width * depth) + threads.y - 1) / threads.y, (C + threads.z - 1) / threads.z);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(attn_weight.type(), "av_fw_cu", 
    ([&] {
        av_fw_kernel<scalar_t><<<blocks, threads>>>(
            attn_weight.packed_accessor<scalar_t, 4, torch::RestrictPtrTraits, size_t>(),
            values.packed_accessor<scalar_t, 4, torch::RestrictPtrTraits, size_t>(),
            output.packed_accessor<scalar_t, 4, torch::RestrictPtrTraits, size_t>(),
            height,
            width,
            depth,
            kernel_size
        );
    }));

    return output;
}